#include "hip/hip_runtime.h"
#include "gpa.cuh"
#include <cstdio>


#include <iostream>
#include <tuple>
#include <fstream>
#include <set>
#include <string>
#include <sstream>
#include <vector>
#include <map>
#include <cassert>
#include <chrono>

#include "sbv_gpu.cuh"
// #include "test.cuh"


#define DEBUG_LOG(x) do { if (debug) std::cout << x << std::endl; } while(0)

#define MAX_EDGE_BUFFER_SIZE 4096



// int test(){
	
// 	sbvInitUnitTest();
// 	sbvSetUnitTest();
// 	sbvContainUnitTest();
// 	sbvToArrayUnitTest();

// 	return 0;

// }

struct PointerSolver{


};

struct PointsToGraph{
	std::map<u_int32_t, std::map<std::string, std::set<u_int32_t>>> graph;
	u_int32_t nodeNum;
	u_int32_t size;
	std::map<u_int32_t, std::set<u_int32_t>> ptrIdToDerivedMrIdsMap;
	std::map<u_int32_t, std::tuple<std::string, u_int32_t, u_int32_t>> nodeId2mrIdAndVerMap;
	PointerSolver solver;


	void printRealPtg(){
		// print points-to set
		std::cout << "printing points-to set.\nMax id is: " << nodeNum << "\nTotal entries: " << size << "\n";
		std::vector<std::string> edgeType{"p", "c", "s", "l", "p_d", "p_u", "p_c", "d", "u", "n", "k", "a"};
		for(auto et : edgeType){
			for(u_int32_t i = 0; i < nodeNum; ++i){
				if(graph.count(i) && graph.at(i).count(et)){
					std::cout << i << " == " << et << " ==> {";
					for(auto pointee : graph.at(i).at(et)){
						std::cout << pointee << " ";
					}
					std::cout << "}\n";
				}
			}
		}
	}
	void printNodeId2mrIdAndVerMap(){
		std::cout << "Printing nodeId2mrIdAndVerMap.\n";
		for(auto p : nodeId2mrIdAndVerMap){
			auto nodeId = p.first;
      auto [funName, mrId, mrVer] = p.second;
      std::cout << nodeId << " ==> " << funName << ":MR_" << mrId << "V_" << mrVer << "\n";
    }
	}
	void printPtrIdToDerivedMrIdsMap(){
		std::cout << "Printing ptrIdToDerivedMrIdsMap.\n";
		for(auto p : ptrIdToDerivedMrIdsMap){
        auto nodeId = p.first;
        for(auto mrId : p.second){
            std::cout << "Node id " << nodeId << " => " << "MR id " << mrId << "\n";
        }
    }
	}

};

PointsToGraph readPtg(std::string fileName){
	std::ifstream inFile(fileName+"-initialPtg.txt");
	if(!inFile){
		std::cerr << "Cannot open file " << fileName << "-initialPtg.txt\n";
		std::terminate();
	}

	PointsToGraph res;
	std::map<u_int32_t, std::map<std::string, std::set<u_int32_t>>> data;
	u_int32_t nodeNum = 0;
	u_int32_t size = 0;


	std::string line;
	while(std::getline(inFile, line)){
		std::stringstream ss(line);
		std::string element;
		std::vector<std::string> elements;
		while(std::getline(ss, element, ',')){
			elements.push_back(element);
		}
		u_int32_t from = static_cast<uint32_t>(std::stoul(elements[0]));
		u_int32_t to = static_cast<uint32_t>(std::stoul(elements[1]));
		nodeNum = std::max(nodeNum, from);
		nodeNum = std::max(nodeNum, to);
		data[from][elements[2]].insert(to);
		++size;
	}

	res.graph = data;
	res.nodeNum = nodeNum;
	res.size = size;

	inFile.close();


	std::map<u_int32_t, std::set<u_int32_t>> ptrIdToDerivedMrIdsMap;
	std::ifstream ptrIdToDerivedMrIdsMapInFile(fileName+"-nodeId2RelatedMrId.txt");
	if(!ptrIdToDerivedMrIdsMapInFile){
		std::cerr << "Cannot open file " << fileName << "-nodeId2RelatedMrId.txt\n";
		std::terminate();
	}

	while(std::getline(ptrIdToDerivedMrIdsMapInFile, line)){
		std::stringstream ss(line);
		std::string element;
		std::vector<std::string> elements;
		while(std::getline(ss, element, ',')){
			elements.push_back(element);
		}
		u_int32_t ptrId = static_cast<uint32_t>(std::stoul(elements[0]));
		std::set<u_int32_t> mrIds;
		for(int i = 1; i < elements.size(); ++i){
			mrIds.insert(static_cast<uint32_t>(std::stoul(elements[i])));
		}
		
		ptrIdToDerivedMrIdsMap.emplace(ptrId, mrIds);
	}

	res.ptrIdToDerivedMrIdsMap = ptrIdToDerivedMrIdsMap;
	ptrIdToDerivedMrIdsMapInFile.close();


	std::map<u_int32_t, std::tuple<std::string, u_int32_t, u_int32_t>> nodeId2mrIdAndVerMap;
	std::ifstream nodeId2mrIdAndVerMapInFile(fileName+"-nodeId2MrId.txt");
	if(!nodeId2mrIdAndVerMapInFile){
		std::cerr << "Cannot open file " << fileName << "-nodeId2MrId.txt\n";
		std::terminate();
	}

	while(std::getline(nodeId2mrIdAndVerMapInFile, line)){
		std::stringstream ss(line);
		std::string element;
		std::vector<std::string> elements;
		while(std::getline(ss, element, ',')){
			elements.push_back(element);
		}

		u_int32_t nodeId = static_cast<uint32_t>(std::stoul(elements[0]));
		uint32_t mrId = static_cast<uint32_t>(std::stoul(elements[2]));
		uint32_t verId = static_cast<uint32_t>(std::stoul(elements[3]));

		std::tuple<std::string, u_int32_t, u_int32_t> t(elements[1], mrId, verId);
		nodeId2mrIdAndVerMap.emplace(nodeId, t);
	}

	res.nodeId2mrIdAndVerMap = nodeId2mrIdAndVerMap;
	nodeId2mrIdAndVerMapInFile.close();

	return res;
}


std::map<std::string, std::vector<SBVCPU*>> preprocessPtgForGpu(PointsToGraph &ptg){
	auto graph = ptg.graph;
	std::vector<std::string> edgeType{"p", "c", "s", "l", "p_d", "p_u", "p_c", "d", "u", "n", "k", "a"};

	std::map<std::string, std::vector<SBVCPU*>> totalSbvs;
	for(auto et : edgeType){
		std::vector<SBVCPU*> sbvs;
		// 0-th entry is intentially introduced because the nodeId starts from 1.
		for(int i = 0; i <= ptg.nodeNum; ++i){
			sbvs.push_back(new SBVCPU());
		}
		totalSbvs.emplace(et, sbvs);
	}

	for(auto pair : graph){
		auto from = pair.first;
		for(auto typeAndTos : pair.second){
			auto type = typeAndTos.first;
			for(auto to : typeAndTos.second){
				totalSbvs[type][from]->setCPU(to);
			}
		}
	}

	return totalSbvs;
}

std::map<std::string, std::vector<SBVCPU*>> preprocessPtgForGpuWithReverseEdges(PointsToGraph &ptg){
	auto graph = ptg.graph;
	std::set<std::string> forwardEdgeType{"p", "u", "s", "p_c", "p_u", "u", "a", "k"};
	std::set<std::string> reverseEdgeType{"c", "l", "d", "p_d", "k", "n"};


	std::map<std::string, std::vector<SBVCPU*>> totalSbvs;
	for(auto et : forwardEdgeType){
		std::vector<SBVCPU*> sbvs;
		// 0-th entry is intentially introduced because the nodeId starts from 1.
		for(int i = 0; i <= ptg.nodeNum; ++i){
			sbvs.push_back(new SBVCPU());
		}
		totalSbvs.emplace(et, sbvs);
	}

	for(auto et : reverseEdgeType){
		std::vector<SBVCPU*> sbvs;
		// 0-th entry is intentially introduced because the nodeId starts from 1.
		for(int i = 0; i <= ptg.nodeNum; ++i){
			sbvs.push_back(new SBVCPU());
		}
		totalSbvs.emplace(et+"-1", sbvs);
	}

	for(auto pair : graph){
		auto from = pair.first;
		for(auto typeAndTos : pair.second){
			auto type = typeAndTos.first;

			if(type == "k"){
				for(auto to : typeAndTos.second){
					totalSbvs[type][from]->setCPU(to);
					totalSbvs[type+"-1"][to]->setCPU(from);
				}
			}
			else if(forwardEdgeType.count(type)){
				for(auto to : typeAndTos.second){
					totalSbvs[type][from]->setCPU(to);
				}
			}
			else{
				for(auto to : typeAndTos.second){
					totalSbvs[type+"-1"][to]->setCPU(from);
				}
			}
			
		}
	}

	// for(auto p : totalSbvs){
	// 	std::cout << p.first << "\n";
	// }

	return totalSbvs;
}



SBV* gpuMallocSbvFromHost(SBV* pool, u_int32_t* pool_ptr) {
    SBV* result = pool + (*pool_ptr);  // just pointer math, no dereferencing
    (*pool_ptr)++;
    return result;
}

__device__ u_int32_t getMrIdOfPtgNode(u_int32_t node, const u_int32_t* nodeIdToMrId, u_int32_t maxNodeId){
		if(node > maxNodeId){
			return 0;
		} 
    return nodeIdToMrId[node];
}

__device__ bool isaVersionofPtr(u_int32_t ptrId, u_int32_t mrAndVerId, const u_int32_t* nodeIdToMrId, u_int32_t maxNodeId,
    const u_int32_t* derived_ids, const u_int32_t* derived_offsets, const u_int32_t* derived_counts){
	
			// ptrid 17 meAndVerId 65

	u_int32_t mrId = getMrIdOfPtgNode(mrAndVerId, nodeIdToMrId, maxNodeId);
	u_int32_t offset = derived_offsets[ptrId];
	u_int32_t count = derived_counts[ptrId];

	// printf("%d %d %d %d %d\n", mrId, offset, count, ptrId, mrAndVerId);

	for (u_int32_t i = 0; i < count; ++i) {
			if (derived_ids[offset + i] == mrId) {
					return true;
			}
	}
	return false;
}

__global__ void rule1(SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *cData, u_int32_t nodeNum, SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// valid id range from [1, nodeNum]
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE cAddr = cData[idx];
	SBV_ADDR_TYPE pAddr = pData[idx];
	if(cAddr == NULL_ADDR || pAddr == NULL_ADDR){
		return;
	}

	SBV *cDstSbv = &sbvPool[cAddr];
	SBV *pDstSbv = &sbvPool[pAddr];
	if(cDstSbv->empty(sbvPool) || pDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t cbuffer[MAX_EDGE_BUFFER_SIZE];
	int clen = cDstSbv->toArray(cbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
	int plen = pDstSbv->toArray(pbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < clen; ++i){
		auto cDst = cbuffer[i];
		SBV_ADDR_TYPE pUpdateAddr = pData[cDst];
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];
			// printf("trying Add p-edge (%d,%d) %d\n", cDst, pDst, idx);
			if(sbvPool[pUpdateAddr].set(pDst, pUpdateAddr, sbvPool, sbvPoolIndex)){

				printf("Adding p-edge (%d,%d)\n", cDst, pDst);
				atomicExch(changed, 1);
			}
		}
	}
}

__global__ void rule2(SBV_ADDR_TYPE *uData, SBV_ADDR_TYPE *lData, SBV_ADDR_TYPE *pcData, SBV_ADDR_TYPE *cData, u_int32_t nodeNum, 
										SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE uAddr = uData[idx];
	SBV_ADDR_TYPE lAddr = lData[idx];

	if(uAddr == NULL_ADDR || lAddr == NULL_ADDR){
		return;
	}


	SBV *uDstSbv = &sbvPool[uAddr];
	SBV *lDstSbv = &sbvPool[lAddr];
	if(uDstSbv->empty(sbvPool) || lDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t ubuffer[MAX_EDGE_BUFFER_SIZE];
	int ulen = uDstSbv->toArray(ubuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t lbuffer[MAX_EDGE_BUFFER_SIZE];
	int llen = lDstSbv->toArray(lbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < ulen; ++i){
		auto uDst = ubuffer[i];
		SBV_ADDR_TYPE pcDstAddr = pcData[uDst];
		if(sbvPool[pcDstAddr].empty(sbvPool)){
			continue;
		}
		SBV_ADDR_TYPE cUpdateAddr = cData[uDst];
		for(int j = 0; j < llen; ++j){
			auto lDst = lbuffer[j];
			if(sbvPool[pcDstAddr].test(lDst, pcDstAddr, sbvPool)){
				if(sbvPool[cUpdateAddr].set(lDst, cUpdateAddr, sbvPool, sbvPoolIndex)){
					printf("Adding c-edge (%d,%d)\n", uDst, lDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
	return;
}

__global__ void rule3(SBV_ADDR_TYPE *dData, SBV_ADDR_TYPE *sData, SBV_ADDR_TYPE *pcData, SBV_ADDR_TYPE *cData, u_int32_t nodeNum, 
										SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE dAddr = dData[idx];
	SBV_ADDR_TYPE sAddr = sData[idx];

	if(dAddr == NULL_ADDR || sAddr == NULL_ADDR){
		return;
	}

	SBV *dDstSbv = &sbvPool[dAddr];
	SBV *sDstSbv = &sbvPool[sAddr];
	if(dDstSbv->empty(sbvPool) || sDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t sbuffer[MAX_EDGE_BUFFER_SIZE];
	int slen = sDstSbv->toArray(sbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t dbuffer[MAX_EDGE_BUFFER_SIZE];
	int dlen = dDstSbv->toArray(dbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < slen; ++i){
		auto sDst = sbuffer[i];
		 
		SBV_ADDR_TYPE pcDstAddr = pcData[sDst];
		if(sbvPool[pcDstAddr].empty(sbvPool)){
			continue;
		}

		
		SBV_ADDR_TYPE cUpdateAddr = cData[sDst];
		for(int j = 0; j < dlen; ++j){
			auto dDst = dbuffer[j];
			if(sbvPool[pcDstAddr].test(dDst, pcDstAddr, sbvPool)){
				if(sbvPool[cUpdateAddr].set(dDst, cUpdateAddr, sbvPool, sbvPoolIndex)){
					printf("Adding c-edge (%d,%d)\n", sDst, dDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
}

__global__ void rule4(SBV_ADDR_TYPE *pdData, SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *dData, u_int32_t nodeNum, SBV *sbvPool, 
					u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId,
    			const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, const u_int32_t* derivedCounts){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE pdAddr = pdData[idx];
	SBV_ADDR_TYPE pAddr = pData[idx];
	if(pdAddr == NULL_ADDR || pAddr == NULL_ADDR){
		return;
	}

	SBV *pdDstSbv = &sbvPool[pdAddr];
	SBV *pDstSbv = &sbvPool[pAddr];
	if(pdDstSbv->empty(sbvPool) || pDstSbv->empty(sbvPool)){
		return;
	}

	SBV_ADDR_TYPE dDstAddr = dData[idx];
	u_int32_t pdbuffer[MAX_EDGE_BUFFER_SIZE];
	int pdlen = pdDstSbv->toArray(pdbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
	int plen = pDstSbv->toArray(pbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < pdlen; ++i){
		auto pdDst = pdbuffer[i];
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];
			if(isaVersionofPtr(pDst, pdDst, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts)){
				if(sbvPool[dDstAddr].set(pdDst, dDstAddr, sbvPool, sbvPoolIndex)){
					printf("Adding d-edge (%d,%d)\n", idx, pdDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
	return;
}

__global__ void rule5(SBV_ADDR_TYPE *puData, SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *uData, u_int32_t nodeNum, SBV *sbvPool, 
					u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId,
    			const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, const u_int32_t* derivedCounts){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE puDstAddr = puData[idx];
	SBV_ADDR_TYPE pDstAddr = pData[idx];

	if(puDstAddr == NULL_ADDR || pDstAddr == NULL_ADDR){
		return;
	}


	SBV *puDstSbv = &sbvPool[puDstAddr];
	SBV *pDstSbv = &sbvPool[pDstAddr];
	if(puDstSbv->empty(sbvPool) || pDstSbv->empty(sbvPool)){
		return;
	}

	SBV_ADDR_TYPE uDstAddr = uData[idx];
	u_int32_t pubuffer[MAX_EDGE_BUFFER_SIZE];
	int pulen = puDstSbv->toArray(pubuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
	int plen = pDstSbv->toArray(pbuffer,sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < pulen; ++i){
		auto puDst = pubuffer[i];
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];
			// printf("(%d,%d,%d)\n", pDst, puDst, isaVersionofPtr(pDst, puDst, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts));
			if(isaVersionofPtr(pDst, puDst, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts)){
				if(sbvPool[uDstAddr].set(puDst, uDstAddr, sbvPool, sbvPoolIndex)){
					printf("Adding u-edge (%d,%d)\n", idx, puDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
}

__global__ void rule6(SBV_ADDR_TYPE *dData, SBV_ADDR_TYPE *kData, SBV_ADDR_TYPE *aData, u_int32_t nodeNum, SBV *sbvPool, 
					u_int32_t *sbvPoolIndex, int *changed){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}



	SBV_ADDR_TYPE dDstAddr = dData[idx];
	SBV *dDstSbv = &sbvPool[dDstAddr];
	if(dDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t dbuffer[MAX_EDGE_BUFFER_SIZE];
	int dlen = dDstSbv->toArray(dbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	for(int i = 0; i < dlen; ++i){
		auto dDst = dbuffer[i];
		
		SBV_ADDR_TYPE kDstAddr = kData[dDst];
		SBV *kDstSbv = &sbvPool[kDstAddr];
		if(kDstSbv->empty(sbvPool)){
			continue;
		}

		u_int32_t kbuffer[MAX_EDGE_BUFFER_SIZE];
		int klen = kDstSbv->toArray(kbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
		for(int j = 0; j < klen; ++j){
			auto kDst = kbuffer[j];
			
			SBV_ADDR_TYPE aDstAddr = aData[kDst];
			SBV *aDstSbv = &sbvPool[aDstAddr];
			if(aDstSbv->set(dDst, aDstAddr, sbvPool, sbvPoolIndex)){
				printf("Adding a-edge (%d,%d)\n", kDst, dDst);
				atomicExch(changed, 1);
			}
		}
	}
}

__global__ void rule7(SBV_ADDR_TYPE *nData, SBV_ADDR_TYPE *kData, SBV_ADDR_TYPE *aData, SBV_ADDR_TYPE *cData, u_int32_t nodeNum, 
					SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE nDstAddr = nData[idx];
	if(nDstAddr == NULL_ADDR){
		return;
	}
	SBV *nDstSbv = &sbvPool[nDstAddr];
	if(nDstSbv->empty(sbvPool)){
		return;
	}


	SBV_ADDR_TYPE cDstAddr = cData[idx];
	const u_int32_t selfMrId = getMrIdOfPtgNode(idx, nodeIdToMrId, nodeNum);

	u_int32_t nbuffer[MAX_EDGE_BUFFER_SIZE];
	int nlen = nDstSbv->toArray(nbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	for(int i = 0; i < nlen; ++i){
		auto nDst = nbuffer[i];
		
		SBV_ADDR_TYPE kDstAddr = kData[nDst];
		SBV *kDstSbv = &sbvPool[kDstAddr];
		if(kDstSbv->empty(sbvPool)){
			continue;
		}
		u_int32_t kbuffer[MAX_EDGE_BUFFER_SIZE];
		int klen = kDstSbv->toArray(kbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
		for(int j = 0; j < klen; ++j){
			auto kDst = kbuffer[j];
			SBV_ADDR_TYPE aDstAddr = aData[kDst];
			SBV *aDstSbv = &sbvPool[aDstAddr];
			if(aDstSbv->empty(sbvPool)){
				continue;
			}
			u_int32_t abuffer[MAX_EDGE_BUFFER_SIZE];
			int alen = aDstSbv->toArray(abuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
			for(int k = 0; k < alen; ++k){
				auto aDst = abuffer[k];
				if(selfMrId != getMrIdOfPtgNode(aDst, nodeIdToMrId, nodeNum)){
					if(sbvPool[cDstAddr].set(nDst, cDstAddr, sbvPool, sbvPoolIndex)){
						printf("Adding c-edge (%d,%d)\n", idx, nDst);
						atomicExch(changed, 1);
					}
				}
			}
		}
	}
}

void solve(std::map<std::string, SBV_ADDR_TYPE*> dataMap, u_int32_t nodeNum, SBV *deviceSbvPool, u_int32_t *deviceSbvPoolIndex,
						const u_int32_t* nodeIdToMrId, const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, 
						const u_int32_t* derivedCounts, bool debug = false, size_t threadsPerBlock = 16){

	int hostChanged = 1;
	int *deviceChanged;
	hipMalloc(&deviceChanged, sizeof(int));

	size_t round = 0;
	// valid node id start from 1.
	size_t blockPerGrid = (nodeNum+threadsPerBlock) / threadsPerBlock;
	std::cout << blockPerGrid << " " << threadsPerBlock << "\n";
	while(hostChanged){
		++round;
		
		hostChanged = 0;
		hipMemcpy(deviceChanged, &hostChanged, sizeof(int), hipMemcpyHostToDevice);

		rule1<<<blockPerGrid,threadsPerBlock>>>(dataMap["p"], dataMap["c"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		CUDA_CHECK(hipGetLastError());
		rule2<<<blockPerGrid,threadsPerBlock>>>(dataMap["u"], dataMap["l"], dataMap["p_c"], dataMap["c"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		CUDA_CHECK(hipGetLastError());
		rule3<<<blockPerGrid,threadsPerBlock>>>(dataMap["d"], dataMap["s"], dataMap["p_c"], dataMap["c"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		CUDA_CHECK(hipGetLastError());
		rule4<<<blockPerGrid,threadsPerBlock>>>(dataMap["p_d"], dataMap["p"], dataMap["d"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId, derivedIds,
										derivedOffsets, derivedCounts);
		CUDA_CHECK(hipGetLastError());
		rule5<<<blockPerGrid,threadsPerBlock>>>(dataMap["p_u"], dataMap["p"], dataMap["u"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId, derivedIds,
										derivedOffsets, derivedCounts);
		CUDA_CHECK(hipGetLastError());
		rule6<<<blockPerGrid,threadsPerBlock>>>(dataMap["d"], dataMap["k"], dataMap["a"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		CUDA_CHECK(hipGetLastError());
		rule7<<<blockPerGrid,threadsPerBlock>>>(dataMap["n"], dataMap["k"], dataMap["a"], dataMap["c"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId);

		CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipMemcpy(&hostChanged, deviceChanged, sizeof(int), hipMemcpyDeviceToHost));
	}
	DEBUG_LOG("Round: " << round);

}

__global__ void rule1Reverse(SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *cDataReverse, u_int32_t nodeNum, SBV *sbvPool, 
					u_int32_t *sbvPoolIndex, int *changed){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// valid id range from [1, nodeNum]
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE cDstAddr = cDataReverse[idx];
	if(cDstAddr == NULL_ADDR){
		return;
	}

	SBV *cDstSbv = &sbvPool[cDstAddr];
	if(cDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t cbuffer[MAX_EDGE_BUFFER_SIZE];
	int clen = cDstSbv->toArray(cbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	SBV_ADDR_TYPE pUpdateAddr = pData[idx];
	if(pUpdateAddr == NULL_ADDR){
		return;
	}
	SBV *pUpdateSbv = &sbvPool[pUpdateAddr];

	for(int i = 0; i < clen; ++i){
		auto cDst = cbuffer[i];

		SBV_ADDR_TYPE pDstAddr = pData[cDst];
		if(pDstAddr == NULL_ADDR){
			return;
		}
		SBV *pDstSbv = &sbvPool[pDstAddr];
		u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
		int plen = pDstSbv->toArray(pbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];
			// printf("trying Add p-edge (%d,%d) %d\n", cDst, pDst, idx);
			if(pUpdateSbv->set(pDst, pUpdateAddr, sbvPool, sbvPoolIndex)){
				// printf("Adding p-edge (%d,%d)\n", idx, pDst);
				atomicExch(changed, 1);
			}
		}
	}
}

__global__ void rule2Reverse(SBV_ADDR_TYPE *uData, SBV_ADDR_TYPE *lDataReverse, SBV_ADDR_TYPE *pcData, SBV_ADDR_TYPE *cDataReverse, 
											u_int32_t nodeNum, SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE lDstAddr = lDataReverse[idx];
	if(lDstAddr == NULL_ADDR){
		return;
	}
	SBV *lDstSbv = &sbvPool[lDstAddr];
	if(lDstSbv->empty(sbvPool)){
		return;
	}


	SBV_ADDR_TYPE cUpdateAddr = cDataReverse[idx];
	u_int32_t lbuffer[MAX_EDGE_BUFFER_SIZE];
	int llen = lDstSbv->toArray(lbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < llen; ++i){

		auto lDst = lbuffer[i];
		SBV_ADDR_TYPE uDstAddr = uData[lDst];
		u_int32_t ubuffer[MAX_EDGE_BUFFER_SIZE];
		int ulen = sbvPool[uDstAddr].toArray(ubuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

		for(int j = 0; j < ulen; ++j){
			auto uDst = ubuffer[j];
			SBV_ADDR_TYPE pcDstAddr = pcData[uDst];
			if(sbvPool[pcDstAddr].test(idx, pcDstAddr, sbvPool)){
				if(sbvPool[cUpdateAddr].set(uDst, cUpdateAddr, sbvPool, sbvPoolIndex)){
					// printf("Adding reverse c-edge (%d,%d)\n", idx, uDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
	return;
}

__global__ void rule3Reverse(SBV_ADDR_TYPE *dDataReverse, SBV_ADDR_TYPE *sData, SBV_ADDR_TYPE *pcData, SBV_ADDR_TYPE *cDataReverse, 
					u_int32_t nodeNum, SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE dDstAddr = dDataReverse[idx];
	if(dDstAddr == NULL_ADDR){
		return;
	}

	SBV *dDstSbv = &sbvPool[dDstAddr];
	if(dDstSbv->empty(sbvPool)){
		return;
	}

	SBV_ADDR_TYPE cUpdateAddr = cDataReverse[idx];
	u_int32_t dbuffer[MAX_EDGE_BUFFER_SIZE];
	int dlen = dDstSbv->toArray(dbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < dlen; ++i){
		auto dDst = dbuffer[i];
		SBV_ADDR_TYPE sDstAddr = sData[dDst];
		u_int32_t sbuffer[MAX_EDGE_BUFFER_SIZE];
		int slen = sbvPool[sDstAddr].toArray(sbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

		for(int j = 0; j < slen; ++j){
			auto sDst = sbuffer[j];

			SBV_ADDR_TYPE pcDstAddr = pcData[sDst];
			if(sbvPool[pcDstAddr].test(idx, pcDstAddr, sbvPool)){
				if(sbvPool[cUpdateAddr].set(sDst, cUpdateAddr, sbvPool, sbvPoolIndex)){
					// printf("Adding reverse c-edge (%d,%d)\n", idx, sDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
}

__global__ void rule4Reverse(SBV_ADDR_TYPE *pdDataReverse, SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *dDataReverse, u_int32_t nodeNum, 
					SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId,
    		const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, const u_int32_t* derivedCounts){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}
	
	SBV_ADDR_TYPE pdDstAddr = pdDataReverse[idx];
	if(pdDstAddr == NULL_ADDR){
		return;
	}
	SBV *pdDstSbv = &sbvPool[pdDstAddr];
	
	if(pdDstSbv->empty(sbvPool)){
		return;
	}

	SBV_ADDR_TYPE dDstAddr = dDataReverse[idx];
	SBV *dDstSbv = &sbvPool[dDstAddr];
	u_int32_t pdbuffer[MAX_EDGE_BUFFER_SIZE];
	int pdlen = pdDstSbv->toArray(pdbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	

	for(int i = 0; i < pdlen; ++i){
		auto pdDst = pdbuffer[i];
		SBV_ADDR_TYPE pDstAddr = pData[pdDst];
		u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
		int plen = sbvPool[pDstAddr].toArray(pbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];

			if(isaVersionofPtr(pDst, idx, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts)){
				if(dDstSbv->set(pdDst, dDstAddr, sbvPool, sbvPoolIndex)){
					// printf("Adding reverse d-edge (%d,%d)\n", idx, pdDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
	return;
}

__global__ void rule5Reverse(SBV_ADDR_TYPE *puData, SBV_ADDR_TYPE *pData, SBV_ADDR_TYPE *uData, u_int32_t nodeNum, SBV *sbvPool, 
					u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId,
    			const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, const u_int32_t* derivedCounts){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}


	SBV_ADDR_TYPE puDstAddr = puData[idx];
	SBV_ADDR_TYPE pDstAddr = pData[idx];
	if(puDstAddr == NULL_ADDR || pDstAddr == NULL_ADDR){
		return;
	}

	SBV *puDstSbv = &sbvPool[puDstAddr];
	SBV *pDstSbv = &sbvPool[pDstAddr];

	if(puDstSbv->empty(sbvPool) || pDstSbv->empty(sbvPool)){
		return;
	}

	SBV_ADDR_TYPE uDstAddr = uData[idx];
	u_int32_t pubuffer[MAX_EDGE_BUFFER_SIZE];
	int pulen = puDstSbv->toArray(pubuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t pbuffer[MAX_EDGE_BUFFER_SIZE];
	int plen = pDstSbv->toArray(pbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < pulen; ++i){
		auto puDst = pubuffer[i];
		for(int j = 0; j < plen; ++j){
			auto pDst = pbuffer[j];
			// printf("(%d,%d,%d)\n", pDst, puDst, isaVersionofPtr(pDst, puDst, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts));
			if(isaVersionofPtr(pDst, puDst, nodeIdToMrId, nodeNum, derivedIds, derivedOffsets, derivedCounts)){
				if(sbvPool[uDstAddr].set(puDst, uDstAddr, sbvPool, sbvPoolIndex)){
					// printf("Adding u-edge (%d,%d)\n", idx, puDst);
					atomicExch(changed, 1);
				}
			}
		}
	}
}

__global__ void rule6Reverse(SBV_ADDR_TYPE *dDataReverse, SBV_ADDR_TYPE *kDataReverse, SBV_ADDR_TYPE *aData, u_int32_t nodeNum, 
					SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}
	
	SBV_ADDR_TYPE kDstAddr = kDataReverse[idx];
	SBV_ADDR_TYPE aDstAddr = aData[idx];
	// printf("kdstsbv %d", kDstSbv->empty());

	SBV *kDstSbv = &sbvPool[kDstAddr];
	if(kDstSbv->empty(sbvPool)){
		return;
	}

	u_int32_t kbuffer[MAX_EDGE_BUFFER_SIZE];
	int klen = kDstSbv->toArray(kbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	for(int i = 0; i < klen; ++i){
		auto kDst = kbuffer[i];

		SBV_ADDR_TYPE dDstAddr = dDataReverse[kDst];
		SBV *dDstSbv = &sbvPool[dDstAddr];
		// printf("dDstSbv %d", dDstSbv->empty());

		if(!dDstSbv->empty(sbvPool)){
			if(sbvPool[aDstAddr].set(kDst, aDstAddr, sbvPool, sbvPoolIndex)){
				// printf("Adding a-edge (%d,%d)\n", idx, kDst);
				atomicExch(changed, 1);
			}
		}
	}
}

__global__ void rule7Reverse(SBV_ADDR_TYPE *nDataReverse, SBV_ADDR_TYPE *kData, SBV_ADDR_TYPE *aData, SBV_ADDR_TYPE *cDataReverse, 
					u_int32_t nodeNum, SBV *sbvPool, u_int32_t *sbvPoolIndex, int *changed, const u_int32_t* nodeIdToMrId){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx == 0 || idx > nodeNum){
	  return;
	}

	SBV_ADDR_TYPE nDstAddr = nDataReverse[idx];
	SBV_ADDR_TYPE kDstAddr = kData[idx];

	SBV *nDstSbv = &sbvPool[nDstAddr];
	SBV *kDstSbv = &sbvPool[kDstAddr];
	if(nDstSbv->empty(sbvPool) || kDstSbv->empty(sbvPool)){
		return;
	}
	SBV_ADDR_TYPE cDstAddr = cDataReverse[idx];
	const u_int32_t selfMrId = getMrIdOfPtgNode(idx, nodeIdToMrId, nodeNum);

	u_int32_t nbuffer[MAX_EDGE_BUFFER_SIZE];
	int nlen = nDstSbv->toArray(nbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
	u_int32_t kbuffer[MAX_EDGE_BUFFER_SIZE];
	int klen = kDstSbv->toArray(kbuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);

	for(int i = 0; i < nlen; ++i){
		auto nDst = nbuffer[i];

		for(int j = 0; j < klen; ++j){
			auto kDst = kbuffer[j];

			SBV_ADDR_TYPE aDstAddr = aData[kDst];
			u_int32_t abuffer[MAX_EDGE_BUFFER_SIZE];
			int alen = sbvPool[aDstAddr].toArray(abuffer, sbvPool, MAX_EDGE_BUFFER_SIZE);
			for(int k = 0; k < alen; ++k){
				auto aDst = abuffer[k];
				if(getMrIdOfPtgNode(nDst, nodeIdToMrId, nodeNum) != getMrIdOfPtgNode(aDst, nodeIdToMrId, nodeNum)){
					if(sbvPool[cDstAddr].set(nDst, cDstAddr, sbvPool, sbvPoolIndex)){
						// printf("Adding reverse c-edge (%d,%d)\n", idx, nDst);
						atomicExch(changed, 1);
					}
				}
			}
		}
	}
}


void solveReverse(std::map<std::string, SBV_ADDR_TYPE*> dataMap, u_int32_t nodeNum, SBV *deviceSbvPool, u_int32_t *deviceSbvPoolIndex,
						const u_int32_t* nodeIdToMrId, const u_int32_t* derivedIds, const u_int32_t* derivedOffsets, 
						const u_int32_t* derivedCounts, bool debug = false, size_t threadsPerBlock = 16){

	DEBUG_LOG("Starting solving with reverse edges\n");

	int hostChanged = 1;
	int *deviceChanged;
	hipMalloc(&deviceChanged, sizeof(int));

	size_t round = 0;
	// valid node id start from 1.
	size_t blockPerGrid = (nodeNum+threadsPerBlock) / threadsPerBlock;
	std::cout << blockPerGrid << " " << threadsPerBlock << "\n";
	while(hostChanged){
		++round;
		// DEBUG_LOG("Round: " << round);
		
		hostChanged = 0;
		CUDA_CHECK(hipMemcpy(deviceChanged, &hostChanged, sizeof(int), hipMemcpyHostToDevice));

		rule1Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["p"], dataMap["c-1"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error1: " << hipGetErrorString(err) << std::endl;
		}
		rule2Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["u"], dataMap["l-1"], dataMap["p_c"], dataMap["c-1"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error2: " << hipGetErrorString(err) << std::endl;
		}
		rule3Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["d-1"], dataMap["s"], dataMap["p_c"], dataMap["c-1"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error3: " << hipGetErrorString(err) << std::endl;
		}
		rule4Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["p_d-1"], dataMap["p"], dataMap["d-1"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId, derivedIds,
										derivedOffsets, derivedCounts);
		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error4: " << hipGetErrorString(err) << std::endl;
		}
		rule5Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["p_u"], dataMap["p"], dataMap["u"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId, derivedIds,
										derivedOffsets, derivedCounts);
		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error5: " << hipGetErrorString(err) << std::endl;
		}
		rule6Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["d-1"], dataMap["k-1"], dataMap["a"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged);
		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error6: " << hipGetErrorString(err) << std::endl;
		}
		rule7Reverse<<<blockPerGrid,threadsPerBlock>>>(dataMap["n-1"], dataMap["k"], dataMap["a"], dataMap["c-1"], nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceChanged, nodeIdToMrId);

		err = hipDeviceSynchronize();
		if (err != hipSuccess) {
				std::cerr << "CUDA kernel error7: " << hipGetErrorString(err) << std::endl;
		}
    CUDA_CHECK(hipMemcpy(&hostChanged, deviceChanged, sizeof(int), hipMemcpyDeviceToHost));
	}
	
	DEBUG_LOG("Round: " << round);
	

}


void verifyResult(std::map<std::string, std::map<u_int32_t, std::set<u_int32_t>>> &edgeTypeToSbvGroupMap, std::string expectedOutputFileName){

	std::ifstream inFile(expectedOutputFileName+"-reversed.out");
	if(!inFile){
		std::cerr << "Cannot open file " << expectedOutputFileName << ".out\n";
		std::terminate();
	}

	std::set<std::tuple<u_int32_t, u_int32_t, std::string>> data;
	std::string line;
	while(std::getline(inFile, line)){
		std::stringstream ss(line);
		std::string element;
		std::vector<std::string> elements;
		while(std::getline(ss, element, ',')){
			elements.push_back(element);
		}
		u_int32_t from = static_cast<uint32_t>(std::stoul(elements[0]));
		u_int32_t to = static_cast<uint32_t>(std::stoul(elements[1]));
		std::tuple<u_int32_t, u_int32_t, std::string> t(from, to, elements[2]);
		data.insert(t);
	}

	inFile.close();

	std::set<std::tuple<u_int32_t, u_int32_t, std::string>> extra;
	// std::vector<std::string> edgeType{"p", "c", "s", "l", "p_d", "p_u", "p_c", "d", "u", "n", "k", "a"};
	// reverse edges
	std::set<std::string> edgeType{"p", "u", "s", "p_c", "p_u", "u", "a", "k", "c-1", "l-1", "d-1", "p_d-1", "n-1"};
	// size_t numberOfTotalEdges = 0;
	for(auto et : edgeType){
		if(!edgeTypeToSbvGroupMap.count(et)){
			continue;
		}
		auto &pts = edgeTypeToSbvGroupMap.at(et);
		for(const auto &[from, tos]: pts){
			for(auto to : tos){
				// check existance for edge from - et -> to in data
				std::tuple<u_int32_t, u_int32_t, std::string> t(from, to, et);
				// ++numberOfTotalEdges;
				auto iter = data.find(t);
				if(iter != data.end()){
          data.erase(iter);
        }
        else{
          extra.insert(t);
        }
			}
		}
	}


	bool error = false;
  if(!data.empty()){
    std::cerr << "ptgSbv missing following points-to entries:\n";
    for(auto [from, to, type] : data){
			// if(type != "p"){
			// 	continue;
			// }
      std::string fromName;
      std::stringstream ss(fromName); 
      ss << from;
      ss << " == " << type << " ==> ";
      ss << to;
      std::cerr << ss.str() << "\n";
    }
    error = true;
  }

  if(!extra.empty()){
    // std::cerr << "ptgSbv contains following extra entries:\n";
    // for(auto [from, to, type] : extra){
    //   std::string fromName;
    //   std::stringstream ss(fromName); 
    //   ss << from;
    //   ss << " == " << type << " ==> ";
    //   ss << to;
    //   std::cerr << ss.str() << "\n";
    // }
    error = true;
  }

  if(error){
    std::cerr << "Fail to verify. " << data.size() << " edges are missing, " <<  extra.size() << " extra edges\n";
  }
  else{
    std::cout << "Verify succeed.\n";
  }

	// std::cout << "Total contains " << numberOfTotalEdges << " edges\n";

}


int gpamain(const std::string ptgFileName){

	auto ptg = readPtg(ptgFileName);
	std::cout << "Initial ptg contains " << ptg.size << " edges\n";
	// auto edgeTypeToSbvGroupMap = preprocessPtgForGpu(ptg);	
	auto edgeTypeToSbvGroupMap = preprocessPtgForGpuWithReverseEdges(ptg);	

	auto start = std::chrono::high_resolution_clock::now();
	std::cout << "1\n";

	// set up gpu memory pool.
	SBV* deviceSbvPool;
	CUDA_CHECK(hipMalloc(&deviceSbvPool, sizeof(SBV) * MAX_ALLOCATED_SBV_NUM));
	u_int32_t index = 0;
	std::cout << "2\n";
	
	// copy initialized SBVs from CPU to GPU.
	std::map<std::string, SBV_ADDR_TYPE*> dataMap;
	std::set<std::string> edgeType{"p", "u", "s", "p_c", "p_u", "u", "a", "k", "c-1", "l-1", "d-1", "p_d-1", "k-1", "n-1"};
	// node 0 is reserved for nullptr. Valid node index starts from 1.
	size_t allocatedCount = ptg.nodeNum+1;
	std::cout << "3\n";

	size_t used = 0;
	for(auto et : edgeType){
		const auto& hostHeads = edgeTypeToSbvGroupMap[et];

		size_t totalBlocks = 0;
    for(size_t i = 0; i < allocatedCount; ++i){
			for(auto* p = hostHeads[i]; p; p = p->next){
				++totalBlocks;
			}
		}
         
		std::vector<SBV> h_blocks; 
		h_blocks.reserve(totalBlocks);
    std::vector<SBV_ADDR_TYPE> head_idx(allocatedCount, -1);

		for(size_t i = 0; i < allocatedCount; ++i) {
			auto* p = hostHeads[i];
			SBV_ADDR_TYPE prev = -1;
			while(p){
				SBV node;
				node.base = p->base;
				for(int w = 0; w < 29; ++w){
					node.bits[w] = p->bits[w];
				} 
				node.next = -1;

				SBV_ADDR_TYPE idx = static_cast<SBV_ADDR_TYPE>(h_blocks.size());
				h_blocks.emplace_back(node);
				if(prev == -1){
					head_idx[i] = idx;
				} 
				else{
					h_blocks[prev].next = idx;
				}           
				prev = idx; 
				p = p->next;
			}
    }

		for (auto& n : h_blocks) {
			if(n.next != -1){
				n.next += static_cast<SBV_ADDR_TYPE>(used);
			} 
    }
    for (auto& h : head_idx) {
			if(h != -1){
				h += static_cast<SBV_ADDR_TYPE>(used);
			} 
    }

		if(!h_blocks.empty()) {
      CUDA_CHECK(hipMemcpy(deviceSbvPool + used, h_blocks.data(), h_blocks.size() * sizeof(SBV), hipMemcpyHostToDevice));
    }

    // std::vector<SBV*> h_head_ptrs(allocatedCount, nullptr);
    // for (size_t i = 0; i < allocatedCount; ++i) {
    //     if (h_heads[i] != -1) h_head_ptrs[i] = deviceSbvPool + h_heads[i];
    // }

    SBV_ADDR_TYPE* d_head_idx = nullptr;  
    CUDA_CHECK(hipMalloc(&d_head_idx, allocatedCount * sizeof(SBV_ADDR_TYPE)));
    CUDA_CHECK(hipMemcpy(d_head_idx, head_idx.data(), allocatedCount * sizeof(SBV_ADDR_TYPE), hipMemcpyHostToDevice));

    dataMap[et] = d_head_idx;

    // Advance global offset
    used += h_blocks.size();

	}

	// CUDA_CHECK(hipMemcpy(d_sbvPoolIndex, &used, sizeof(uint32_t), hipMemcpyHostToDevice));







	std::cout << "4\n";

	u_int32_t* deviceSbvPoolIndex;
	CUDA_CHECK(hipMalloc(&deviceSbvPoolIndex, sizeof(u_int32_t)));
	// index refers to the address of the first available SBV block in the GPU pool after initialization.
	CUDA_CHECK(hipMemcpy(deviceSbvPoolIndex, &used, sizeof(uint32_t), hipMemcpyHostToDevice));

	std::vector<u_int32_t> hostNodeIdToMrId(allocatedCount, 0);
	for (const auto &[nodeId, tup] : ptg.nodeId2mrIdAndVerMap){
    hostNodeIdToMrId[nodeId] = std::get<1>(tup);  // the mrId
	}

	u_int32_t* deviceNodeIdToMrId;
	CUDA_CHECK(hipMalloc(&deviceNodeIdToMrId, hostNodeIdToMrId.size() * sizeof(u_int32_t)));
	CUDA_CHECK(hipMemcpy(deviceNodeIdToMrId, hostNodeIdToMrId.data(), allocatedCount * sizeof(u_int32_t), hipMemcpyHostToDevice));

	std::cout << "5\n";

	std::vector<u_int32_t> hostDerivedIds, hostDerivedOffsets, hostDerivedCounts;

	for (u_int32_t i = 0; i < allocatedCount; ++i){
			auto it = ptg.ptrIdToDerivedMrIdsMap.find(i);
			if(it == ptg.ptrIdToDerivedMrIdsMap.end()){
					hostDerivedOffsets.push_back(hostDerivedIds.size());
					hostDerivedCounts.push_back(0);
			} 
			else{
				hostDerivedOffsets.push_back(hostDerivedIds.size());
				hostDerivedCounts.push_back(it->second.size());
				for(u_int32_t id : it->second) {
						hostDerivedIds.push_back(id);
				}
			}
	}

	std::cout << "6\n";
	u_int32_t *deviceDerivedIds, *deviceDerivedOffsets, *deviceDerivedCounts;

	CUDA_CHECK(hipMalloc(&deviceDerivedIds, hostDerivedIds.size() * sizeof(u_int32_t)));
	CUDA_CHECK(hipMalloc(&deviceDerivedOffsets, hostDerivedOffsets.size() * sizeof(u_int32_t)));
	CUDA_CHECK(hipMalloc(&deviceDerivedCounts, hostDerivedCounts.size() * sizeof(u_int32_t)));
	CUDA_CHECK(hipMemcpy(deviceDerivedIds, hostDerivedIds.data(), hostDerivedIds.size() * sizeof(u_int32_t), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceDerivedOffsets, hostDerivedOffsets.data(), hostDerivedOffsets.size() * sizeof(u_int32_t), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceDerivedCounts, hostDerivedCounts.data(), hostDerivedCounts.size() * sizeof(u_int32_t), hipMemcpyHostToDevice));

	hipDeviceSynchronize();



	auto preprocessed = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(preprocessed - start);
  std::cout << "Preprocessing time: " << duration.count() << " ms\n";

	hipEvent_t cudastart, cudastop;
	hipEventCreate(&cudastart);
	hipEventCreate(&cudastop);

	hipEventRecord(cudastart);

	// solve(dataMap, ptg.nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceNodeIdToMrId, deviceDerivedIds, deviceDerivedOffsets,
	// 			deviceDerivedCounts, true, 1);

	solveReverse(dataMap, ptg.nodeNum, deviceSbvPool, deviceSbvPoolIndex, deviceNodeIdToMrId, deviceDerivedIds, deviceDerivedOffsets,
				deviceDerivedCounts, true);

	hipEventRecord(cudastop);

	hipEventSynchronize(cudastop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, cudastart, cudastop);

	std::cout << "Solving time (GPU only): " << milliseconds << " ms\n";

	hipEventDestroy(cudastart);
	hipEventDestroy(cudastop);

	hipDeviceSynchronize();
	auto solved = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds>(solved - preprocessed);
  std::cout << "Solving time: " << duration.count() << " ms\n";

	auto end = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
  std::cout << "Total time: " << duration.count() << " ms\n";

	u_int32_t totalSbvBlockNum;
	CUDA_CHECK(hipMemcpy(&totalSbvBlockNum, deviceSbvPoolIndex, sizeof(u_int32_t), hipMemcpyDeviceToHost));

	std::cout << "Total number of sbv block allocated is " << totalSbvBlockNum << "\n";








	std::map<std::string, std::map<u_int32_t, std::set<u_int32_t>>> edgeResultMap;
	std::vector<SBV> hostPool;
	hostPool.resize(MAX_ALLOCATED_SBV_NUM);
	CUDA_CHECK(hipMemcpy(hostPool.data(), deviceSbvPool, MAX_ALLOCATED_SBV_NUM * sizeof(SBV), hipMemcpyDeviceToHost));



	// for(const std::string& label : edgeType) {
  //   SBV_ADDR_TYPE *deviceLabelData = dataMap[label];

  //   // Host array of device pointers
  //   std::vector<SBV_ADDR_TYPE> hostSbvPtrArray(allocatedCount);
  //   CUDA_CHECK(hipMemcpy(hostSbvPtrArray.data(), deviceLabelData, allocatedCount * sizeof(SBV_ADDR_TYPE), hipMemcpyDeviceToHost));

  //   for(u_int32_t src = 0; src < allocatedCount; ++src){
	// 		SBV_ADDR_TYPE deviceAddr = hostSbvPtrArray[src];
	// 		if(deviceAddr == NULL_ADDR){
	// 			continue;
	// 		} 

			// Copy chain of SBVs to host
			// std::vector<SBV> chain;
			// SBV curr;
			// CUDA_CHECK(hipMemcpy(&curr, deviceSbv, sizeof(SBV), hipMemcpyDeviceToHost));
			// chain.push_back(curr);
			// while (curr.next) {
			// 		CUDA_CHECK(hipMemcpy(&curr, curr.next, sizeof(SBV), hipMemcpyDeviceToHost));
			// 		chain.push_back(curr);
			// }

			// Decode all IDs
	// 		SBV_ADDR_TYPE curr = deviceAddr;
	// 		while(curr != NULL_ADDR){
	// 			SBV &sbv = hostPool[curr];
	// 			for(int i = 0; i < WORDS_PER_BLOCK; ++i){
	// 				u_int32_t word = sbv.bits[i];
	// 				if(!word){
	// 					continue;
	// 				} 
	// 				for(int bit = 0; bit < 32; ++bit){
	// 					if(word & (1u << bit)){
	// 						u_int32_t baseId = sbv.base;
	// 						u_int32_t wordId = 28 - i;
	// 						u_int32_t inWordId = bit;
	// 						u_int32_t id = baseId * 928 + wordId * 32 + inWordId;
	// 						edgeResultMap[label][src].insert(id);
	// 					}
	// 				}
	// 			}

	// 			curr = sbv.next;
	// 		}
  //   }
	// }

	// verifyResult(edgeResultMap, ptgFileName);

	// for(const auto &[label, pts] : edgeResultMap){
	// 	if(label != "p"){
	// 		continue;
	// 	}
  //   for(const auto &[src, dsts] : pts){
	// 		if(dsts.empty()){
	// 			continue;
	// 		}
	// 		std::cout << src << " ==> {";
	// 		for(auto dst : dsts){
	// 			std::cout << dst << " ";
	// 		}
	// 		std::cout << "}\n";
  //   }
	// }

	// clean up

	std::cout << "7\n";
	CUDA_CHECK(hipFree(deviceSbvPool));
	CUDA_CHECK(hipFree(deviceSbvPoolIndex));
	CUDA_CHECK(hipFree(deviceNodeIdToMrId));
	CUDA_CHECK(hipFree(deviceDerivedIds));
	CUDA_CHECK(hipFree(deviceDerivedOffsets));
	CUDA_CHECK(hipFree(deviceDerivedCounts));

	std::cout << "8\n";


	// for(const auto &[et, ptrs] : dataMap){
	// 	CUDA_CHECK(hipFree(ptrs));
	// }
	
	std::cout << "9\n";


	for(auto et : edgeType){
		for(auto sbv : edgeTypeToSbvGroupMap[et]){
			delete sbv;
		}
	}

	std::cout << "10\n";


	return 0;
	
}