#include "hip/hip_runtime.h"
#include "sbv_gpu.cuh"


__device__ __host__ void SBV::init(u_int32_t baseVal){
		base = baseVal;
		#pragma unroll
		for (int i = 0; i < WORDS_PER_BLOCK; ++i)
				bits[i] = 0;
		next = NULL_ADDR;
}

__device__ __host__ bool SBV::isValidIndex(u_int32_t id){
	return id > 0 && id < WORDS_PER_BLOCK * 32 * UINT32_MAX;
}

__device__ bool SBV::set(u_int32_t id, u_int64_t selfId, SBV *d_sbv_pool, u_int32_t *d_sbv_pool_ptr){
	
	if(!isValidIndex(id)){
		return false;
	}

	

	const u_int32_t baseId = id / 928;
	const u_int32_t bitsId = id % 928;
	const u_int32_t wordId = bitsId / 32;
	const u_int32_t inWordId = bitsId % 32;
	const u_int32_t mask = 1u << inWordId;

	u_int64_t curr = selfId;
	u_int64_t prev = NULL_ADDR;

	while(curr != NULL_ADDR && d_sbv_pool[curr].base < baseId){
		prev = curr;
		curr = d_sbv_pool[curr].next;
	}

	if(curr != NULL_ADDR && d_sbv_pool[curr].base == baseId){
		u_int32_t old = d_sbv_pool[curr].bits[28-wordId];
		atomicOr(&d_sbv_pool[curr].bits[28 - wordId], mask);
		// curr->bits[28-wordId] |= mask;
		return old != d_sbv_pool[curr].bits[28-wordId];
	}
	else{
		u_int64_t newIdx = gpu_malloc_sbv(d_sbv_pool, d_sbv_pool_ptr);
		SBV &newBlock = d_sbv_pool[newIdx];
		newBlock.base = baseId;
		#pragma unroll
		for(int i = 0; i < WORDS_PER_BLOCK; ++i){
			newBlock.bits[i] = 0;
		}
		// newBlock->bits[28 - wordId] |= mask;
		atomicOr(&newBlock.bits[28 - wordId], mask);
		newBlock.next = curr;

		if(prev != NULL_ADDR){
			d_sbv_pool[prev].next = newIdx;
		}
		else{
			u_int64_t backupIdx = gpu_malloc_sbv(d_sbv_pool, d_sbv_pool_ptr);
			SBV &backup = d_sbv_pool[backupIdx];
			SBV &head = d_sbv_pool[selfId];

			backup.base = head.base;
			#pragma unroll
			for(int i = 0; i < WORDS_PER_BLOCK; ++i){
				backup.bits[i] = head.bits[i];
			}
			backup.next = head.next;
			head.base = baseId;
			#pragma unroll
			for(int i = 0; i < WORDS_PER_BLOCK; ++i){
				head.bits[i] = 0;
			}
			// bits[28 - wordId] |= mask;
			atomicOr(&head.bits[28 - wordId], mask);
			head.next = backupIdx;
		}

		// unlockSBV(&this->lock);
		return true;
	}
}

__device__ bool SBV::test(u_int32_t id, u_int64_t selfId, SBV *d_sbv_pool){
	if(!isValidIndex(id)){
		return false;
	}

	u_int32_t baseId = id / 928;
	u_int32_t bitsId = id % 928;
	u_int32_t wordId = bitsId / 32;
	u_int32_t inWordId = bitsId % 32;

	u_int64_t curr = selfId;
	while(curr != NULL_ADDR){
		const SBV &node = d_sbv_pool[curr];
		if(node.base == baseId){
			return (node.bits[28-wordId] >> inWordId) & 1u;
		}
		if(node.base > baseId){
			break;
		}
		curr = node.next;
	}

	return false;
}

// __host__ __device__ bool emptyBlock() {
// 	#pragma unroll
// 	for (int i = 0; i < WORDS_PER_BLOCK; ++i) {
// 			// keep your original 28 - i order
// 			if (bits[28 - i] != 0u) return false;
// 	}
// 	return true;
// }

__device__ __host__ bool SBV::empty(SBV *d_sbv_pool) const{

	u_int64_t curr = static_cast<u_int64_t>(this - d_sbv_pool);

	while (curr != NULL_ADDR) {
		const SBV& node = d_sbv_pool[curr];
		for (u_int32_t i = 0; i < WORDS_PER_BLOCK; ++i) {
			if (node.bits[WORDS_PER_BLOCK - 1 - i] != 0) {
				return false;
			}
		}
		curr = node.next;
	}
	return true;

}

__device__ size_t SBV::toArray(uint32_t* out, SBV *d_sbv_pool, u_int32_t max_len = 4096){


	size_t count = 0;
	u_int64_t curr = static_cast<u_int64_t>(this - d_sbv_pool);
	

	while(curr != NULL_ADDR && count < max_len) {
		SBV &node = d_sbv_pool[curr];

		#pragma unroll
		for(int i = 0; i < WORDS_PER_BLOCK && count < max_len; ++i){
			uint32_t word = node.bits[WORDS_PER_BLOCK-1-i];
			if(word == 0){
				continue;
			}
			
			for(u_int32_t j = 0; j < 32 && count < max_len; ++j){
				if((word >> j) & 1u){
					out[count++] = node.base * 928 + i * 32 + j;
				}
			}
		}
		curr = node.next;
	}
	return count;  // number of valid elements in out[]
}

__host__	bool SBV::setCPU(u_int32_t id, u_int64_t selfId, std::vector<SBV>& pool){

	bool changed = false;

	if(!isValidIndex(id)){
		return changed;
	}

	u_int32_t baseId = id / 928;
	u_int32_t bitsId = id % 928;
	u_int32_t wordId = bitsId / 32;
	u_int32_t inWordId = bitsId % 32;


	u_int64_t curr = selfId;
	u_int64_t prev = NULL_ADDR;

	while(curr != NULL_ADDR && pool[curr].base < baseId){
		prev = curr;
		curr = pool[curr].next;
	}

	if(curr != NULL_ADDR && pool[curr].base == baseId){
		uint32_t old = pool[curr].bits[28-wordId];
		pool[curr].bits[28-wordId] |= (1u << inWordId);
		if(old != pool[curr].bits[28-wordId]){
			changed = true;
		}
	}
	else{
		// add new block
		const u_int64_t newIdx = static_cast<u_int64_t>(pool.size());
		SBV newSbv;
		newSbv.base = baseId;
		for(int i = 0; i < WORDS_PER_BLOCK; ++i){
			newSbv.bits[i] = 0u;
		} 
		newSbv.bits[28-wordId] |= (1u << inWordId);
		newSbv.next = curr;
		pool.emplace_back(newSbv);

		if(prev != NULL_ADDR){
			pool[prev].next = newIdx;
		}
		else{
			const u_int64_t backupIdx = static_cast<u_int64_t>(pool.size());
			pool.emplace_back(pool[selfId]);
			SBV &head = pool[selfId];
			head.base = baseId;
			for(int i = 0; i < WORDS_PER_BLOCK; ++i){
				head.bits[i] = 0;
			}
			head.bits[28-wordId] |= (1u << inWordId);
			head.next = backupIdx;
		}
		changed = true;
	}
	return changed;
}

__host__ void SBV::print(const std::vector<SBV>& pool, u_int64_t headId) const{
	
	u_int64_t curr = headId;
	std::cout << "{";
	while(curr != NULL_ADDR){
		const SBV &node = pool[curr];
		u_int32_t id = node.base * 928;
		for(u_int32_t i = 0; i < WORDS_PER_BLOCK; ++i){
			if(node.bits[28-i] == 0){
				continue;
			}
			for(u_int32_t j = 0; j < 32; ++j){
				if((node.bits[28-i] >> j) & 1u){
					std::cout << id + i*32 + j << " ";
				}
			}
		}
		curr = node.next;

	}
	std::cout << "}\n";
}





__device__ u_int64_t gpu_malloc_sbv(SBV* pool, u_int32_t* pool_ptr){
	assert(*pool_ptr < MAX_ALLOCATED_SBV_NUM && "Max number of node exceeded.");
	uint32_t idx = atomicAdd(pool_ptr, 1u);
  return static_cast<u_int64_t>(idx);
}



__device__ __host__ void SBVCPU::init(u_int32_t baseVal){
	base = baseVal;
	for(int i = 0; i < WORDS_PER_BLOCK; ++i)
		bits[i] = 0;
	next = nullptr;
}



// todo : check git log and copy the cpy version of sbv
__device__ __host__ bool SBVCPU::isValidIndex(u_int32_t id){
        return id > 0 && id < 928 * UINT32_MAX;
}

// __device__ bool SBVCPU::set(u_int32_t id, SBV *d_sbv_pool, u_int32_t *d_sbv_pool_ptr){

//         // lockSBV(&this->lock);

//         if(!isValidIndex(id)){
//                 return false;
//         }



//         const u_int32_t baseId = id / 928;
//         const u_int32_t bitsId = id % 928;
//         const u_int32_t wordId = bitsId / 32;
//         const u_int32_t inWordId = bitsId % 32;
//         const u_int32_t mask = 1u << inWordId;

//         SBVCPU *curr = this;
//         SBVCPU *prev = nullptr;

//         while(curr && curr->base < baseId){
//                 prev = curr;
//                 curr = curr->next;

//         }

//         if(curr && curr->base == baseId){
//                 u_int32_t old = curr->bits[28-wordId];
//                 atomicOr(&curr->bits[28 - wordId], mask);
//                 // curr->bits[28-wordId] |= mask;
//                 // unlockSBV(&this->lock);
//                 return old != curr->bits[28-wordId];
//         }
//         else{
//                 SBVCPU* newBlock = gpu_malloc_sbv(d_sbv_pool, d_sbv_pool_ptr);
//                 newBlock->base = baseId;
//                 for(int i = 0; i < WORDS_PER_BLOCK; ++i){
//                         newBlock->bits[i] = 0;
//                 }
//                 // newBlock->bits[28 - wordId] |= mask;
//                 atomicOr(&newBlock->bits[28 - wordId], mask);
//                 newBlock->next = curr;

//                 if(prev){
//                         prev->next = newBlock;
//                 }
//                 else{
//                         SBVCPU *backup = gpu_malloc_sbv(d_sbv_pool, d_sbv_pool_ptr);
//                         backup->base = base;
//                         for(int i = 0; i < WORDS_PER_BLOCK; ++i){
//                                 backup->bits[i] = bits[i];
//                         }
//                         backup->next = next;
//                         this->base = baseId;
//                         for(int i = 0; i < 29; ++i){
//                                 this->bits[i] = 0;
//                         }
//                         // bits[28 - wordId] |= mask;
//                         atomicOr(&bits[28 - wordId], mask);
//                         this->next = backup;
//                 }

//                 // unlockSBV(&this->lock);
//                 return true;
//         }
// }

__device__ bool SBVCPU::test(u_int32_t id){
        if(!isValidIndex(id)){
                return false;
        }

        u_int32_t baseId = id / 928;
        u_int32_t bitsId = id % 928;
        u_int32_t wordId = bitsId / 32;
        u_int32_t inWordId = bitsId % 32;

        const SBVCPU *curr = this;
        while(curr){
                if(curr->base == baseId){
                        return (curr->bits[28-wordId] >> inWordId) & 1u;
                }
                if(curr->base > baseId){
                        break;
                }
                curr = curr->next;
        }

        return false;
}

__device__ __host__ bool SBVCPU::empty() const{

        const SBVCPU* curr = this;
        while (curr) {
                for (u_int32_t i = 0; i < 29; ++i) {
                        if (curr->bits[28 - i] != 0) {
                                return false;
                        }
                }
                curr = curr->next;
        }
        return true;

}

__device__ int SBVCPU::toArray(uint32_t* out, u_int32_t max_len = 4096){
        int count = 0;
        SBVCPU* curr = this;

        while (curr) {
                for (int i = 0; i < WORDS_PER_BLOCK && count < max_len; ++i) {
                        if(curr->bits[28-i] == 0){
                                continue;
                        }
                        uint32_t word = curr->bits[28-i];
                        for(u_int32_t j = 0; j < 32 && count < max_len; ++j){
                                if((word >> j) & 1u){
                                        out[count++] = curr->base * 928 + i * 32 + j;
                                }
                        }
                }
                curr = curr->next;
        }
        return count;  // number of valid elements in out[]
}

__host__        bool SBVCPU::setCPU(u_int32_t id){

        bool changed = false;

        if(!isValidIndex(id)){
                return changed;
        }

        u_int32_t baseId = id / 928;
        u_int32_t bitsId = id % 928;
        u_int32_t wordId = bitsId / 32;
        u_int32_t inWordId = bitsId % 32;

        SBVCPU *curr = this;
        SBVCPU *prev = nullptr;

        while(curr && curr->base < baseId){
                prev = curr;
                curr = curr->next;
        }

        if(curr && curr->base == baseId){
                uint32_t old = curr->bits[28-wordId];
                curr->bits[28-wordId] |= (1u << inWordId);
                if(old != curr->bits[28-wordId]){
                        changed = true;
                }
        }
        else{
                // add new block
                SBVCPU *newSbv = new SBVCPU();
                newSbv->base = baseId;
                newSbv->bits[28-wordId] |= (1u << inWordId);
                newSbv->next = curr;
                if(prev){
                        prev->next = newSbv;
                }
                else{
                        SBVCPU *backup = new SBVCPU();
                        backup->base = base;
                        for(int i = 0; i < WORDS_PER_BLOCK; ++i){
                                backup->bits[i] = bits[i];
                        }
                        backup->next = next;

                        this->base = baseId;
                        for(int i = 0; i < 29; ++i){
                                this->bits[i] = 0;
                        }
                        this->setCPU(id);
                        this->next = backup;
                }
                changed = true;
        }
        return changed;
}

__host__ void SBVCPU::print() const{

        const SBVCPU *curr = this;
        std::cout << "{";
        while(curr){
                u_int32_t id = curr->base * 928;
                for(u_int32_t i = 0; i < 29; ++i){
                        if(curr->bits[28-i] == 0){
                                continue;
                        }
                        for(u_int32_t j = 0; j < 32; ++j){
                                if((curr->bits[28-i] >> j) & 1u){
                                        std::cout << id + i*32 + j << " ";
                                }
                        }
                }
                curr = curr->next;

        }
        std::cout << "}\n";
}